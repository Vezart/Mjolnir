#include "hip/hip_runtime.h"
//============================================================
//  File:       im_cuda.cu
//  Author:     John Gauch
//  Date:       Summer 2010
//============================================================

#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define PIXEL(x,y) ( (((y)+ydim)%ydim)*xdim + ((x)+xdim)%xdim )

// Global variables for timers
#define CUDA_TIMER
#ifdef CUDA_TIMER
hipEvent_t cuda_start;
hipEvent_t cuda_stop;
float cuda_time;
#endif

//============================================================
void query_cuda()
{
    // Get number of CUDA devices
    int devCount;
    hipGetDeviceCount(&devCount);
    printf("There are %d CUDA devices.\n", devCount);

    // Iterate through CUDA devices
    for (int i = 0; i < devCount; ++i)
    {
        // Get CUDA device properties
        printf("\nCUDA Device #%d\n", i);
        hipDeviceProp_t devProp;
        hipGetDeviceProperties(&devProp, i);

        // Print CUDA device properties
        printf("Major revision number:         %d\n",  (int)devProp.major);
        printf("Minor revision number:         %d\n",  (int)devProp.minor);
        printf("Name:                          %s\n",  devProp.name);
        printf("Total global memory:           %d\n",  (int)devProp.totalGlobalMem);
        printf("Total shared memory per block: %d\n",  (int)devProp.sharedMemPerBlock);
        printf("Total registers per block:     %d\n",  (int)devProp.regsPerBlock);
        printf("Warp size:                     %d\n",  (int)devProp.warpSize);
        printf("Maximum memory pitch:          %d\n",  (int)devProp.memPitch);
        printf("Maximum threads per block:     %d\n",  (int)devProp.maxThreadsPerBlock);
        for (int i = 0; i < 3; ++i)
           printf("Maximum dimension %d of block:  %d\n", i, (int)devProp.maxThreadsDim[i]);
        for (int i = 0; i < 3; ++i)
           printf("Maximum dimension %d of grid:   %d\n", i, (int)devProp.maxGridSize[i]);
        printf("Clock rate:                    %d\n",  (int)devProp.clockRate);
        printf("Total constant memory:         %d\n",  (int)devProp.totalConstMem);
        printf("Texture alignment:             %d\n",  (int)devProp.textureAlignment);
        printf("Concurrent copy and execution: %s\n",  (devProp.deviceOverlap ? "Yes" : "No"));
        printf("Number of multiprocessors:     %d\n",  (int)devProp.multiProcessorCount);
        printf("Kernel execution timeout:      %s\n",  (devProp.kernelExecTimeoutEnabled ? "Yes" : "No"));
    }
}

//============================================================
void create_timer()
{
#ifdef CUDA_TIMER
   hipEventCreate(&cuda_start);
   hipEventCreate(&cuda_stop);
#endif
}

//============================================================
void destroy_timer()
{
#ifdef CUDA_TIMER
   hipEventDestroy(cuda_start);
   hipEventDestroy(cuda_stop);
#endif
}

//============================================================
void start_timer()
{
#ifdef CUDA_TIMER
   hipEventRecord(cuda_start,0);
#endif
}

//============================================================
void end_timer(char *str)
{
#ifdef CUDA_TIMER
   hipEventRecord(cuda_stop,0);
   hipEventSynchronize(cuda_stop);
   hipEventElapsedTime(&cuda_time, cuda_start, cuda_stop);
   printf("%s: %f\n", str, cuda_time/1000);
#endif
}

//============================================================
void cuda_config(int length, int &thread_cnt, int &block_cnt)
{
   thread_cnt = 0;
   block_cnt = 0;
   hipDeviceProp_t devProp;
   if (hipGetDeviceProperties(&devProp, 0) != hipSuccess) return;
   thread_cnt = devProp.maxThreadsPerBlock / 4;
   block_cnt = length / thread_cnt;
   if (block_cnt * thread_cnt < length) block_cnt++;
   // printf("thread_cnt=%d, block_cnt=%d\n", thread_cnt, block_cnt);
}

//============================================================
__global__ void binomial_kernel(float *cu_in, float *cu_out, int xdim, int ydim)
{
   // get (x,y) coordinates
   int pixel = blockIdx.x*blockDim.x + threadIdx.x;
   if (pixel > xdim * ydim) return;
   int x = pixel % xdim;
   int y = pixel / xdim;

   cu_out[PIXEL(x,y)] 
      = (cu_in[PIXEL(x+1,y+1)] 
      + cu_in[PIXEL(x+1,y)] * 2
      + cu_in[PIXEL(x+1,y-1)]
      + cu_in[PIXEL(x,y+1)] * 2
      + cu_in[PIXEL(x,y)] * 4
      + cu_in[PIXEL(x,y-1)] * 2
      + cu_in[PIXEL(x-1,y+1)] 
      + cu_in[PIXEL(x-1,y)] * 2
      + cu_in[PIXEL(x-1,y-1)]) / (float)16.0;
}

//============================================================
float* binomial_cuda(float *data, int count, int xdim, int ydim)
{
   // declare variables
   float *cu_in;  
   float *cu_out;
   int length = xdim*ydim;
   size_t size_in = length*sizeof(float);
   size_t size_out = length*sizeof(float);
   int thread_cnt, block_cnt;
   cuda_config(length, thread_cnt, block_cnt);
   dim3 blocks(block_cnt, 1, 1);
   dim3 threads(thread_cnt, 1, 1);

   // allocate array on device
   create_timer();
   hipMalloc((void **) &cu_in, size_in);
   hipMalloc((void **) &cu_out, size_out);

   // copy data from host to device
   start_timer();
   hipMemcpy(cu_in, data,  size_in, hipMemcpyHostToDevice);
   end_timer("Copy memory to device");

   // run kernel
   start_timer();
   while (count >= 2)
   {
      binomial_kernel <<< blocks, threads >>> (cu_in, cu_out, xdim, ydim);
      binomial_kernel <<< blocks, threads >>> (cu_out, cu_in, xdim, ydim);
      count -= 2;
   }
   if (count == 1)
      binomial_kernel <<< blocks, threads >>> (cu_in, cu_out, xdim, ydim);
   end_timer("Run kernel");

   // copy data from device to host
   start_timer();
   if (count == 1)
      hipMemcpy(data, cu_out,  size_out, hipMemcpyDeviceToHost);
   else
      hipMemcpy(data, cu_in,  size_in, hipMemcpyDeviceToHost);
   end_timer("Copy memory from device");

   // cleanup
   hipFree(cu_in);
   hipFree(cu_out);
   destroy_timer();
   return data;
}

//============================================================
__global__ void binomial_kernel(short *cu_in, short *cu_out, int xdim, int ydim)
{
   // get (x,y) coordinates
   int pixel = blockIdx.x*blockDim.x + threadIdx.x;
   if (pixel > xdim * ydim) return;
   int x = pixel % xdim;
   int y = pixel / xdim;

   cu_out[PIXEL(x,y)] 
      = (cu_in[PIXEL(x+1,y+1)] 
      + cu_in[PIXEL(x+1,y)] * 2
      + cu_in[PIXEL(x+1,y-1)]
      + cu_in[PIXEL(x,y+1)] * 2
      + cu_in[PIXEL(x,y)] * 4
      + cu_in[PIXEL(x,y-1)] * 2
      + cu_in[PIXEL(x-1,y+1)] 
      + cu_in[PIXEL(x-1,y)] * 2
      + cu_in[PIXEL(x-1,y-1)] + 8) / 16;
}

//============================================================
short* binomial_cuda(short *data, int count, int xdim, int ydim)
{
   // declare variables
   short *cu_in;  
   short *cu_out;
   int length = xdim*ydim;
   size_t size_in = length*sizeof(short);
   size_t size_out = length*sizeof(short);
   int thread_cnt, block_cnt;
   cuda_config(length, thread_cnt, block_cnt);
   dim3 blocks(block_cnt, 1, 1);
   dim3 threads(thread_cnt, 1, 1);

   // allocate array on device
   create_timer();
   hipMalloc((void **) &cu_in, size_in);
   hipMalloc((void **) &cu_out, size_out);

   // copy data from host to device
   start_timer();
   hipMemcpy(cu_in, data, size_in, hipMemcpyHostToDevice);
   end_timer("Copy memory to device");

   // run kernel
   start_timer();
   while (count >= 2)
   {
      binomial_kernel <<< blocks, threads >>> (cu_in, cu_out, xdim, ydim);
      binomial_kernel <<< blocks, threads >>> (cu_out, cu_in, xdim, ydim);
      count -= 2;
   }
   if (count == 1)
      binomial_kernel <<< blocks, threads >>> (cu_in, cu_out, xdim, ydim);
   end_timer("Run kernel");

   // copy data from device to host
   start_timer();
   if (count == 1)
      hipMemcpy(data, cu_out, size_out, hipMemcpyDeviceToHost);
   else
      hipMemcpy(data, cu_in, size_in, hipMemcpyDeviceToHost);
   end_timer("Copy memory from device");

   // cleanup
   hipFree(cu_in);
   hipFree(cu_out);
   destroy_timer();
   return data;
}

//============================================================
__global__ void gradient_kernel(float *cu_in, float *cu_out, int xdim, int ydim)
{
   // get (x,y) coordinates
   int pixel = blockIdx.x*blockDim.x + threadIdx.x;
   if (pixel > xdim * ydim) return;
   int x = pixel % xdim;
   int y = pixel / xdim;

   float dx 
      = cu_in[PIXEL(x+1,y+1)] 
      + cu_in[PIXEL(x+1,y)] * 2
      + cu_in[PIXEL(x+1,y-1)]
      - cu_in[PIXEL(x-1,y+1)]
      - cu_in[PIXEL(x-1,y)] * 2
      - cu_in[PIXEL(x-1,y-1)];
   float dy 
      = cu_in[PIXEL(x+1,y+1)] 
      + cu_in[PIXEL(x,y+1)] * 2
      + cu_in[PIXEL(x-1,y+1)]
      - cu_in[PIXEL(x+1,y-1)]
      - cu_in[PIXEL(x,y-1)] * 2
      - cu_in[PIXEL(x-1,y-1)];

   if (dx < 0) dx = -dx;
   if (dy < 0) dy = -dy;
   cu_out[PIXEL(x,y)] = (dx+dy) / (float)8.0;
}

//============================================================
float* gradient_cuda(float *data, int xdim, int ydim)
{
   // declare variables
   float *cu_in;  
   float *cu_out;
   int length = xdim*ydim;
   size_t size_in = length*sizeof(float);
   size_t size_out = length*sizeof(float);
   int thread_cnt, block_cnt;
   cuda_config(length, thread_cnt, block_cnt);
   dim3 blocks(block_cnt, 1, 1);
   dim3 threads(thread_cnt, 1, 1);

   // allocate array on device
   create_timer();
   hipMalloc((void **) &cu_in, size_in);
   hipMalloc((void **) &cu_out, size_out);

   // copy data from host to device
   start_timer();
   hipMemcpy(cu_in, data, size_in, hipMemcpyHostToDevice);
   end_timer("Copy memory to device");

   // run kernel
   start_timer();
   gradient_kernel <<< blocks, threads >>> (cu_in, cu_out, xdim, ydim);
   end_timer("Run kernel");

   // copy data from device to host
   start_timer();
   hipMemcpy(data, cu_out, size_out, hipMemcpyDeviceToHost);
   end_timer("Copy memory from device");

   // cleanup
   hipFree(cu_in);
   hipFree(cu_out);
   return data;
}

//============================================================
__global__ void gradient_kernel(short *cu_in, short *cu_out, int xdim, int ydim)
{
   // get (x,y) coordinates
   int pixel = blockIdx.x*blockDim.x + threadIdx.x;
   if (pixel > xdim * ydim) return;
   int x = pixel % xdim;
   int y = pixel / xdim;

   int dx 
      = cu_in[PIXEL(x+1,y+1)] 
      + cu_in[PIXEL(x+1,y)] * 2
      + cu_in[PIXEL(x+1,y-1)]
      - cu_in[PIXEL(x-1,y+1)]
      - cu_in[PIXEL(x-1,y)] * 2
      - cu_in[PIXEL(x-1,y-1)];
   int dy 
      = cu_in[PIXEL(x+1,y+1)] 
      + cu_in[PIXEL(x,y+1)] * 2
      + cu_in[PIXEL(x-1,y+1)]
      - cu_in[PIXEL(x+1,y-1)]
      - cu_in[PIXEL(x,y-1)] * 2
      - cu_in[PIXEL(x-1,y-1)];

   if (dx < 0) dx = -dx;
   if (dy < 0) dy = -dy;
   cu_out[PIXEL(x,y)] = (dx+dy) / 8;
}

//============================================================
short* gradient_cuda(short *data, int xdim, int ydim)
{
   // declare variables
   short *cu_in;  
   short *cu_out;
   int length = xdim*ydim;
   size_t size_in = length*sizeof(short);
   size_t size_out = length*sizeof(short);
   int thread_cnt, block_cnt;
   cuda_config(length, thread_cnt, block_cnt);
   dim3 blocks(block_cnt, 1, 1);
   dim3 threads(thread_cnt, 1, 1);

   // allocate array on device
   create_timer();
   hipMalloc((void **) &cu_in, size_in);
   hipMalloc((void **) &cu_out, size_out);

   // copy data from host to device
   start_timer();
   hipMemcpy(cu_in, data, size_in, hipMemcpyHostToDevice);
   end_timer("Copy memory to device");

   // run kernel
   start_timer();
   gradient_kernel <<< blocks, threads >>> (cu_in, cu_out, xdim, ydim);
   end_timer("Run kernel");

   // copy data from device to host
   start_timer();
   end_timer("Copy memory from device");
   hipMemcpy(data, cu_out, size_out, hipMemcpyDeviceToHost);

   // cleanup
   hipFree(cu_in);
   hipFree(cu_out);
   return data;
}

